#include "hip/hip_runtime.h"
#include "activation.hpp"

namespace cuANN
{

 __host__ __device__ float sigmoid_func ( const float x )
{
    return 1.0 / (1.0 + exp ( -x ) );
}

__host__ __device__ float fast_sigmoid ( const float x )
{
    return x / ( 1 + abs( x ) );
}

__host__ __device__ float tanh_func ( const float x )
{
    return tanh( x );
}

}

#include "hip/hip_runtime.h"
#include "ann.hpp"

/// This is the implementation cubin, which uses template classes
namespace cuANN
{

__host__ ann::ann (
                    unsigned int input_neurons,
                    unsigned int hidden_neurons,
                    unsigned int hidden_layers,
                    unsigned int output_neurons
                  )
: input_neurons_( input_neurons ), 
  hidden_neurons_( hidden_neurons ), 
  hidden_layers_ ( hidden_layers ), 
  output_neurons_ ( output_neurons )
{
    // If we have hidden neurons - TODO: Update formula without INPUT WEIGHTS
    if ( hidden_neurons_ > 0 )
    {
        per_layer_ = std::ceil( hidden_neurons_ / hidden_layers_ );
        hidden_neurons_ = per_layer_ * hidden_layers_;

        // Hidden weights = ( per_layer^2 * (hidden_layers -1) ) + (per_layer * output)
        //                  the first line calculates weights from input to first hiddden
        //                  the second line calculates weights within hidden layers
        //                  the third line calculates weights from hidden to output neurons
        unsigned int i_w = input_neurons_ * per_layer_;
        unsigned int h_w = std::pow( per_layer_, 2 ) * (hidden_layers_ -1 );
        unsigned int o_w = per_layer_ * output_neurons_ ;

        weights_ = thrust::device_vector<float>( i_w + h_w + o_w );

        // Index the weights appropriately
        w_index_.push_back( std::make_pair( 0, i_w ) );

        for ( int i = 1; i <= hidden_layers_; i++ )
        {
            int h = std::pow( per_layer_, 2 );
            unsigned int k = i * h;
            unsigned int left = weights_.size() - k;
            if ( left >= h )
                w_index_.push_back( std::make_pair( k, k + h ) );

            else
                w_index_.push_back( std::make_pair( k, weights_.size() ) );
        }

    }
    // no hidden neurons
    else
    {
        per_layer_ = 0;
        weights_ = thrust::device_vector<float>( input_neurons_ * output_neurons );
        w_index_.push_back( std::make_pair( 0, weights_.size() ) );
    }
   
    // low and upper random bounds
    float upper = .2f;
    float lower = -.2f;

    thrust::counting_iterator<float> index_sequence_begin(0);
    auto now = std::chrono::system_clock::now();
    auto seed = std::chrono::duration_cast<std::chrono::milliseconds>( now.time_since_epoch()).count();

    // Random Init all hidden weights (regardless of layer index) 
    thrust::transform(  index_sequence_begin,
                        index_sequence_begin + weights_.size(),
                        weights_.begin(),
                        prg( upper, lower, seed ) );

    std::cout << "input neurons: " << input_neurons << std::endl;
    std::cout << "hidden neurons: " << hidden_neurons_ << " (per layer: " << per_layer_ << ")" << std::endl;
    std::cout << "output neurons: " << output_neurons << std::endl;
    std::cout << "weights: " << weights_.size() << std::endl;
}


__host__ float ann::train (
                              const cuANN::data & input,
                              float mse_stop,
                              unsigned int epochs,
                              float learning,
                              float momentum
                          )
{
    // Run epoch iterations, doing a back_propagation
    // after each iteration
    // Stop only if mse_stop is achieved, or if we run out of epochs
    // TODO...
    // Iterate and load all training data (input & output) on the device
    // And re-use it to avoid re-allocations.

    return -1.f;
}

__host__ ann::h_vector ann::propagate ( ann::d_vector input ) const
{
    if ( input.size() != input_neurons_ )
        throw std::runtime_error( "ann::propagate param input size doesn't match input layer size" );

    // NOTE - COMMENTED FOR TESTING - UNCOMMENT !
    // Put the input values Through the Sigmoid Function
    auto dim = dim_find_1D( input.size() );
    float * input_ptr = thrust::raw_pointer_cast( input.data() );
    sigmoid_kernel<<<dim.num_blocks_x,dim.block_threads_x>>>( input_ptr, input.size() );

    // propagate through the first (input to hidden/output)
    // if this is the only layer, this is our only propagation
    thrust::device_vector<float> out = prop_layer( std::get<0>(w_index_[0]),
                                                   std::get<1>(w_index_[0]),
                                                   input );

    // Repeat for hidden layers
    for ( int i = 1; i < w_index_.size(); i++)
    {
        out = prop_layer( std::get<0>(w_index_[i]),
                          std::get<1>(w_index_[i]),
                                             out );
    }

    return out;
}


__host__ float ann::epoch ( 
                                d_vector & input,
                                unsigned int input_len,
                                d_vector & output,
                                unsigned int output_len
                          )
{
    // Accumulate squared errors
    thrust::host_vector<float> errors( input.size() );

    // TODO: Iterate input, vector at specific interval
    // TODO: Extract output, vector at specific interval
    // TODO: Propagate Input, measure Output, 
    //       ^^^ STORE someplace the Resulting vectors: Sum( Input * Weight )
    // NOTE: Propagation during a training epoch is different from simple (TEST) propagation.

    // TODO: Calculate Error at last layer
    // TODO: Working backwards, calculate Error at all layers
    // TODO: Calculate for each Gradient for each weight
    // TODO: Adjust Each weight, using the gradient

    //return sum_errors  / num_errors;
    return 0.f;
}


__host__ ann::d_vector ann::prop_layer ( 
                                          unsigned int weights_begin,
                                          unsigned int weights_end,
                                          const ann::d_vector & input
                                       ) const
{
    // TODO: ADD BIAS NEURON FOR EACH AND EVERY INPUT 
    //       MAKE SURE THAT ONE NEURON FIRES A VALUE OF 1.f
    //       Only way to do this, is to re-alloc (alloc) a new input vector here.

    unsigned int weights_size = weights_end - weights_begin;
    unsigned int w_per_i = weights_size / input.size();

    // Get raw pointers for CUDA kernel
    const float * input_ptr = thrust::raw_pointer_cast( input.data() );
    const float * weight_ptr = thrust::raw_pointer_cast( weights_.data() ) + weights_begin;

    // Allocate Vectorized Matrix and get Raw pointer
    thrust::device_vector<float> mtx_output( weights_size );
    float * mtx_ptr = thrust::raw_pointer_cast( mtx_output.data() );

    // Calculate block theads and block number
    // Our X grid, is the Input size, our Y grid, is the number of weights per Input
    auto dm_a = dim_find_prop_mtx( input.size(), w_per_i );

    // set the threads per block and number of blocks
    dim3 tPb1( dm_a.block_threads_x, dm_a.block_threads_y );
    dim3 nB1( dm_a.num_blocks_x, dm_a.num_blocks_y );

    // Multiply Each Input, with its Row of Weights (Matrix of Weights) resulting in a Matrix of Sums (per Input/Row)
    // This is a 2D Grid iterations, using as input Weights and Inputs, and the Matrix as output
    // Width (Columns) is # weights per input, Height (Rows) is # inputs
    prop_matrix<<<nB1,tPb1>>>( weight_ptr, input_ptr, mtx_ptr, w_per_i, input.size() );

    // Sums Row vector
    thrust::device_vector<float> sums ( w_per_i );
    float * sums_ptr = thrust::raw_pointer_cast( sums.data() );
    auto dm_b = dim_find_1D( w_per_i );

    // Sum each column into a vector row : I[i] * W[i] = I[k]
    // Sumarize Columns, using Matrix as Input, Sums vector as output, 
    // where Width = Weights per I (# of Columns), and Height = # of Inputs (# of Rows )
    sum_columns<<<dm_b.num_blocks_x,dm_b.block_threads_x>>>( mtx_ptr, sums_ptr, w_per_i, input.size() );

    // Finally, run the sums through the activation function
    // Same num of blocks, same number of threads per block
    sigmoid_kernel<<<dm_b.num_blocks_x,dm_b.block_threads_x>>>( sums_ptr, sums.size() ); 
   
    return sums;
}

__host__ ann::d_vector ann::output_errors (
                                              d_vector ideal,
                                              d_vector actual
                                          ) const
{
    if ( ideal.size() != actual.size() )
        throw std::runtime_error ( "cuANN::ann::output_error: ideal vector diff size from actual vector" );

    // NOTE: Running this on GPU using device vectors is probably an overkill.
    //       This should be profiled, and if found to offer no advantage, moved to host code only
    
    // This is PART of the MSE: (Ideal[i] - Actual[i])^2 + ...
    thrust::device_vector<float> errors( ideal.size() );
    float * ideal_ptr = thrust::raw_pointer_cast( ideal.data() );
    float * actual_ptr = thrust::raw_pointer_cast( actual.data() );
    float * errors_ptr = thrust::raw_pointer_cast( errors.data() );

    // First calculate all errors 
    auto dm = dim_find_1D( ideal.size() );
    squared_error<<<dm.num_blocks_x,dm.block_threads_x>>>( ideal_ptr, actual_ptr, errors_ptr );

    // This is how u sum all errors using thrust::reduce
    //float squared_error = thrust::reduce( errors.begin(), errors.end() );
    return errors;
}


};
